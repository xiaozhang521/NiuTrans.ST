#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
* Copyright (C) 2017, Natural Language Processing Lab, Northeastern University.
* All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/*
* $Created by: Lin Ye (email: linye2015@outlook.com) 2018-08-03
* $Update by: Lin Ye (email: linye2015@outlook.com) 2019-07-06 float16/int added
*/

#include "../../XDevice.h"
#include "../../XTensor.h"
#include "../shape/IsSameShaped.h"
#include "Clip.h"
#include "Clip.cuh"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA
/*
set each entry to its clip value (CUDA Kernel)
>> a - pointer to input data array
>> b - pointer to output data array
>> lower - the lower border
>> upper - the upper border
>> size - size of the data array
*/
template <class T>
__global__
void KernelClip(T * a, T * b, T lower, T upper, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size) {
        if (a[i] > upper)
            b[i] = upper;
        else if (a[i] < lower)
            b[i] = lower;
        else
            b[i] = a[i];
    }
}

/*
set each entry to its clip value
>> a - input tensor we are processing
>> b - output tensor we are processing
>> lower - the lower border
>> upper - the upper border
*/
void _CudaClip(const XTensor * a, XTensor * b, DTYPE lower, DTYPE upper)
{
    CheckNTErrors((_IsSameShaped(a, b)), "Input tensors should have the same type!");
    CheckNTErrors((a->isSparse == false), "TODO!");

    int gridSize[3];
    int blockSize[3];

    GDevs.GetCudaThread(a->devID, a->unitNum, gridSize, blockSize);

    dim3 blocks(gridSize[0]);
    dim3 threads(blockSize[0]);

    int devIDBackup;
    ProtectCudaDev(a->devID, devIDBackup);

    if (a->dataType == DEFAULT_DTYPE) {
        KernelClip << <blocks, threads >> >((DTYPE*)a->data, (DTYPE*)b->data, lower, upper, a->unitNum);
    }
    else if (a->dataType == X_FLOAT16) {
#ifdef HALF_PRECISION
        half lower2 = __float2half(lower);
        half upper2 = __float2half(upper);
        KernelClip << <blocks, threads >> >((__half*)a->data, (__half*)b->data, lower2, upper2, a->unitNum);
#else
        ShowNTErrors("Recompile the code with HALF_PRECISION!");
#endif
    }
    else if (a->dataType == X_INT) {
        int lower2 = (int)lower;
        int upper2 = (int)upper;

        KernelClip << <blocks, threads >> >((int *)a->data, (int *)b->data, lower2, upper2, a->unitNum);
    }
    else if (a->dataType == X_INT8) {
        ShowNTErrors("TODO!");
    }
    else {
        ShowNTErrors("TODO!");
    }

    BacktoCudaDev(a->devID, devIDBackup);
}

/*
clip backward computation of dE/dx (Cuda kernel)

dy/dx = 1     if lower <= x <= upper
0     otherwise

>> dedy - dE/dy
>> dedx - dE/dx
>> y - y of the function
>> x - x of the function
>> lower 
>> upper 
*/
__global__
void KernelClipBackward(DTYPE * dedy, DTYPE * dedx, DTYPE * y, DTYPE * x, DTYPE lower, DTYPE upper, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size) {
        DTYPE s = x[i];
        if (s > upper || s < lower)
            dedx[i] = 0;
        else
            dedx[i] = dedy[i];
    }
}

/*
backward computation (Cuda version)

dE/dx = dE/dy * dy/dx

hard tanh: y =  upper    if x > upper
x    if lower <= x <= upper
lower    if x< lower

and dy/dx =  1    if lower <= x <= upper
0    otherwise

>> gold - gold standard to measure error (or loss)
>> y - output of the function
>> x - input of the function
>> dedy - dE/dy
>> dedx - dE/dx
>> lossName - type of loss function, e.g., cross entropy
*/
void _CudaClipBackward(XTensor * y, XTensor * x, XTensor * dedy, XTensor * dedx, DTYPE lower, DTYPE upper)
{
    if (x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE) {

        int gridSize[3], blockSize[3];

        GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

        int devIDBackup;
        ProtectCudaDev(x->devID, devIDBackup);

        /* dE/dx = dE/dy * dy/dx */
        KernelClipBackward <<<dim3(gridSize[0]), dim3(blockSize[0])>>>
                             ((DTYPE*)dedy->data,
                              (DTYPE*)dedx->data,
                              (DTYPE*)y->data, (DTYPE*)x->data,
                              lower, upper,
                              x->unitNum);

        BacktoCudaDev(x->devID, devIDBackup);
    }
    else
        ShowNTErrors("TODO!");
}


#endif // USE_CUDA
} // namespace nts(NiuTrans.Tensor)
