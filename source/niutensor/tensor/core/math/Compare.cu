#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northeastern University.
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: Xu Chen (email: hello_master1954@163.com) 2018-07-31
 */

#include <math.h>
#include "../../XDevice.h"
#include "../../XName.h"
#include "Compare.h"
#include "Compare.cuh"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

__device__
DTYPE cudaIsEqual(DTYPE a, DTYPE b)
{
    return (a == b ? 1.0F : 0.0F);
}

__device__
DTYPE cudaIsNotEqual(DTYPE a, DTYPE b)
{
    return (a != b ? 1.0F : 0.0F);
}

#define SIMPLE_COMPARE_FUNCTION_GPU(funcName, origFunc)                     \
__global__                                                                  \
void Kernel##funcName(DTYPE * a, DTYPE * b, int size, DTYPE number)         \
{                                                                           \
    int i = blockDim.x * blockIdx.x + threadIdx.x;                          \
                                                                            \
    if (i < size)                                                           \
        b[i] = (DTYPE)origFunc(a[i], number);                               \
}                                                                           \
__global__                                                                  \
void Kernel##funcName(__half * a, __half * b, int size, __half number)      \
{                                                                           \
    return;                                                                 \
}                                                                           \
void _Cuda##funcName(const XTensor * a, XTensor * b, DTYPE number)          \
{                                                                           \
                                                                            \
    int gridSize[3];                                                        \
    int blockSize[3];                                                       \
                                                                            \
    GDevs.GetCudaThread(a->devID, a->unitNum, gridSize, blockSize);         \
                                                                            \
    dim3 blocks(gridSize[0]);                                               \
    dim3 threads(blockSize[0]);                                             \
                                                                            \
    int devIDBackup;                                                        \
    ProtectCudaDev(a->devID, devIDBackup);                                  \
                                                                            \
    if (a->dataType == DEFAULT_DTYPE) {                                     \
        Kernel##funcName<<<blocks, threads>>>                               \
                         ((DTYPE*)a->data, (DTYPE*)b->data,                 \
                           a->unitNum, (DTYPE)number);                      \
    }                                                                       \
    else if (a->dataType == X_FLOAT16) {                                    \
        Kernel##funcName<<<blocks, threads>>>                               \
                         ((__half*)a->data, (__half*)b->data,               \
                           a->unitNum, (__half)number);                     \
    }                                                                       \
    else {                                                                  \
        ShowNTErrors("TODO!");                                              \
    }                                                                       \
                                                                            \
    BacktoCudaDev(a->devID, devIDBackup);                                   \
}                                                                           \

SIMPLE_COMPARE_FUNCTION_GPU(Equal, cudaIsEqual)
SIMPLE_COMPARE_FUNCTION_GPU(NotEqual, cudaIsNotEqual)

#define SIMPLE_MAX_MIN_FUNCTION_GPU(funcName, origFunc)                     \
__global__                                                                  \
void Kernel##funcName(DTYPE * a, DTYPE * b, DTYPE * c, int size)            \
{                                                                           \
    int i = blockDim.x * blockIdx.x + threadIdx.x;                          \
                                                                            \
    if (i < size)                                                           \
        c[i] = (DTYPE)origFunc(a[i], b[i]);                                 \
}                                                                           \
__global__                                                                  \
void Kernel##funcName(__half * a, __half * b, __half * c, int size)         \
{                                                                           \
    return;                                                                 \
}                                                                           \
void _Cuda##funcName(const XTensor * a, const XTensor * b, XTensor * c)     \
{                                                                           \
                                                                            \
    int gridSize[3];                                                        \
    int blockSize[3];                                                       \
                                                                            \
    GDevs.GetCudaThread(a->devID, a->unitNum, gridSize, blockSize);         \
                                                                            \
    dim3 blocks(gridSize[0]);                                               \
    dim3 threads(blockSize[0]);                                             \
                                                                            \
    int devIDBackup;                                                        \
    ProtectCudaDev(a->devID, devIDBackup);                                  \
                                                                            \
    if (a->dataType == DEFAULT_DTYPE) {                                     \
        Kernel##funcName<<<blocks, threads>>>                               \
                         ((DTYPE*)a->data, (DTYPE*)b->data,                 \
                          (DTYPE*)c->data, a->unitNum);                     \
    }                                                                       \
    else if (a->dataType == X_FLOAT16) {                                    \
        Kernel##funcName<<<blocks, threads>>>                               \
                         ((__half*)a->data, (__half*)b->data,               \
                          (__half*)c->data, a->unitNum);                    \
    }                                                                       \
    else {                                                                  \
        ShowNTErrors("TODO!");                                              \
    }                                                                       \
                                                                            \
    BacktoCudaDev(a->devID, devIDBackup);                                   \
}    

SIMPLE_MAX_MIN_FUNCTION_GPU(Equal, cudaIsEqual)
SIMPLE_MAX_MIN_FUNCTION_GPU(NotEqual, cudaIsNotEqual)

SIMPLE_MAX_MIN_FUNCTION_GPU(Max, max)
SIMPLE_MAX_MIN_FUNCTION_GPU(Min, min)

#endif // USE_CUDA

} // namespace nts(NiuTrans.Tensor)