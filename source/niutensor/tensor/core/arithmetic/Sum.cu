#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
* Copyright (C) 2017, Natural Language Processing Lab, Northeastern University.
* All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-24
* $Update by: Lin Ye (email: linye2015@outlook.com) 2019-07-02 float16/int/int8 added
*/

#include "../../XDevice.h"
#include "../../XUtility.h"
#include "Sum.cuh"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/*
summation of data arrays (CUDA Kernel)
c = a  + b * \beta
>> a - A matrix
>> b - another matrix
>> c - where we put a+b
>> size - the size of a/b/c
>> beta - the coefficient
*/
template <class T>
__global__
void KernelADD(T * a, T * b, T * c, int size, T beta)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size)
        c[i] = a[i] + b[i] * beta;
}


/*
tensor summation c = a + b * \beta (cuda version)
>> a - a tensor
>> b - another tensor
>> c - where we put a+b*\beta. we save it in a if c is NULL
>> beta - the scaling factor
*/
void _CudaSum(const XTensor * a, const XTensor * b, XTensor * c, DTYPE beta)
{
    CheckNTErrors(a && b && c, "Empty tensor input!");
    CheckNTErrors((a->unitNum == b->unitNum && a->unitNum == c->unitNum),
                  "Unmatched tensors in addition!");
    CheckNTErrors((a->dataType == b->dataType && a->dataType == c->dataType),
                  "Unmatched tensors in addition!");
    CheckNTErrors((a->devID == b->devID && a->devID == c->devID),
                  "The tensors must be on the same!");
    CheckNTErrors((a->dataType == DEFAULT_DTYPE && b->dataType == DEFAULT_DTYPE && c->dataType == DEFAULT_DTYPE) ||
                  (a->dataType == X_FLOAT16 && b->dataType == X_FLOAT16 && c->dataType == X_FLOAT16) ||
                  (a->dataType == X_INT && b->dataType == X_INT && c->dataType == X_INT) ||
                  (a->dataType == X_INT8 && b->dataType == X_INT8 && c->dataType == X_INT8),
                  "The sum function does not support this datatype.");

    int devIDBackup = XDevice::GetGPUDevice();
    XDevice::SetGPUDevice(a->devID);

    if (!a->isSparse && !b->isSparse) {
        CheckNTErrors(!c->isSparse,
            "Illegal use of sparse matrix in addition!");

        if (a->dataType == DEFAULT_DTYPE &&
            b->dataType == DEFAULT_DTYPE &&
            c->dataType == DEFAULT_DTYPE)
        {
            hipblasHandle_t * handle = NULL;
            if ((a->mem != NULL) && (b->mem != NULL)) {
                hipblasHandle_t * handleA = a->mem->GetCublasHandle();
                hipblasHandle_t * handleB = b->mem->GetCublasHandle();
                handle = *handleA != 0 ? handleA : handleB;
            }
            else {
                handle = GDevs.GetCudaHandle(a->devID);
            }

            if ((c == a && handle != NULL) && *handle != 0) {
#ifdef DOUBELPRICSION
                hipblasDaxpy(*handle, a->unitNum, &beta, (DTYPE*)b->data, 1, (DTYPE*)a->data, 1);
#else
                hipblasSaxpy(*handle, a->unitNum, &beta, (DTYPE*)b->data, 1, (DTYPE*)a->data, 1);
#endif
            }
            else {
                int gridSize[3], blockSize[3];

                GDevs.GetCudaThread(a->devID, a->unitNum, gridSize, blockSize);
                dim3 blocks(gridSize[0]);
                dim3 threads(blockSize[0]);

                KernelADD << <blocks, threads >> >((DTYPE*)a->data, (DTYPE*)b->data, (DTYPE*)c->data, a->unitNum, beta);
            }
        }
        else if (a->dataType == X_FLOAT16 &&
                 b->dataType == X_FLOAT16 &&
                 c->dataType == X_FLOAT16)
        {
#ifdef HALF_PRECISION
            int gridSize[3], blockSize[3];

            GDevs.GetCudaThread(a->devID, a->unitNum, gridSize, blockSize);
            dim3 blocks(gridSize[0]);
            dim3 threads(blockSize[0]);

            half beta1 = __float2half(beta);

            KernelADD << <blocks, threads >> >((__half *)a->data, (__half *)b->data, (__half *)c->data, a->unitNum, beta1);
#else
            ShowNTErrors("Recompile the code with HALF_PRECISION!");
#endif
        }
        else if (a->dataType == X_INT &&
                 b->dataType == X_INT &&
                 c->dataType == X_INT)
        {
            int gridSize[3], blockSize[3];

            GDevs.GetCudaThread(a->devID, a->unitNum, gridSize, blockSize);
            dim3 blocks(gridSize[0]);
            dim3 threads(blockSize[0]);
            KernelADD << <blocks, threads >> >((int*)a->data, (int*)b->data, (int*)c->data, a->unitNum, (int)beta);
        }
        else {
            // TODO!!
            ShowNTErrors("TODO!");
        }
    }
    else {
        // TODO!!
        ShowNTErrors("TODO!");
    }

    XDevice::SetGPUDevice(devIDBackup);
}

/* summation over arrays
tensor summation c = a + b * \beta (cuda version) with an input handle
>> devID - device ID (MUST >= 0)
>> handle - cuda handle
>> a - an array
>> b - another array
>> c - where we put a+b
>> size - size of the array
>> beta - the coefficient
*/
void _CudaSumWithHandle(int devID, hipblasHandle_t * handle, DTYPE * a, DTYPE * b, DTYPE * c, int size, DTYPE beta)
{
    if (size == 0)
        return;

    if (c == NULL)
        c = a;

    CheckNTErrors((a && b && c), "Empty arrays in addition!");

    int devIDBackup;
    ProtectCudaDev(devID, devIDBackup);

    if (c == a) {
#ifdef DOUBELPRICSION
        hipblasDaxpy(*handle, size, &beta, b, 1, a, 1);
#else
        hipblasSaxpy(*handle, size, &beta, b, 1, a, 1);
#endif
    }
    else {
        int gridSize[3], blockSize[3];

        GDevs.GetCudaThread(devID, size, gridSize, blockSize);

        dim3 blocks(gridSize[0]);
        dim3 threads(blockSize[0]);

        KernelADD<<<blocks, threads>>>((DTYPE*)a, (DTYPE*)b, (DTYPE*)c, size, beta);
    }

    BacktoCudaDev(devID, devIDBackup);
}

#endif // USE_CUDA

} // namespace nts(NiuTrans.Tensor)
