#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
* Copyright (C) 2018, Natural Language Processing Lab, Northeastern University.
* All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-07-29
* &Updated by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-12-26
* Add summation by broadcasting.
* $Update by: Lin Ye (email: linye2015@outlook.com) 2019-07-24 float16 added
*/

#include "SumDim.cuh"
#include "../../XDevice.h"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/* 
tensor summation of a tensor and a row vector
c = a + b * \beta 
where a is a tensor and b is a row vector
>> a - pointer to the data array of a
>> b - pointer to the data array of b
>> c - pointer to the data array of c
>> rowNum - number of rows of a and c
>> colNum - number of columns of a and c (i.e., the size of b)
>> beta - the scaling factor
*/
template <class T, bool betaFired>
__global__
void KernelAddWithRow(T * a, T * b, T * c, int rowNum, int colNum, T beta)
{
    __shared__ T bv[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if(col >= colNum || row >= rowNum)
        return;

    if(threadIdx.y == 0)
        bv[threadIdx.x] = b[col];

    __syncthreads();

    int offset = colNum * row + col;
    if(betaFired)
        c[offset] = a[offset] + bv[threadIdx.x] * beta;
    else
        c[offset] = a[offset] + bv[threadIdx.x];
}

/* 
tensor summation of a tensor and a colum vector
c = a + b * \beta 
where a is a tensor and b is a colum vector
>> a - pointer to the data array of a
>> b - pointer to the data array of b
>> c - pointer to the data array of c
>> rowNum - number of rows of a and c (i.e., the size of b)
>> colNum - number of columns of a and c 
>> blockNum - size of a block (matrix), i.e., rowNum * colNum
>> blockNum - number of matrics 
>> beta - the scaling factor
*/
template <class T, bool betaFired>
__global__
void KernelAddWithCol(T * a, T * b, T * c, int rowNum, int colNum, int blockSize, int blockNum, T beta)
{
    __shared__ T bv[MAX_CUDA_THREAD_NUM_PER_BLOCK];

    int colIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    int col = colIndex % colNum;
    int block = colIndex / colNum;

    if(row >= rowNum || block >= blockNum)
        return;

    if(threadIdx.x == 0)
        bv[threadIdx.y] = b[row];

    __syncthreads();

    int offset = block * blockSize + row * colNum + col;
    
    if(betaFired)
        c[offset] = a[offset] + bv[threadIdx.y] * beta;
    else
        c[offset] = a[offset] + bv[threadIdx.y];
}

/*
tensor summation (cuda version)

c = a + b * \beta 
where the size of b is equal to the n-th dimension of a, 
i.e., a is summed with b by broadcasting

>> a - a tensor
>> b - another tensor whose size is equal to that of dimension n of a
>> c - where we put a+b*\beta. we save it in a if c is NULL
>> n - the dimension index
>> beta - the scaling factor
*/
void _CudaSumDim(const XTensor * a, const XTensor * b, XTensor * c, int n, DTYPE beta)
{
    CheckNTErrors(a && b && c, "Empty tensor input!");
    CheckNTErrors(a->unitNum == c->unitNum, "Unmatched tensors in addition!");
    CheckNTErrors(a->dataType == b->dataType && a->dataType == c->dataType,
                  "Unmatched data types in addition!");
    CheckNTErrors(a->order == c->order, "The input tensors do not have the same order in addition!");
    CheckNTErrors(!a->isSparse && !b->isSparse && !c->isSparse, "Dense tensors are required!");
    CheckNTErrors(a->dimSize[n] == b->unitNum, "Wrong tensor size!");

    int stride = 1;
    int blockSize = a->dimSize[n];
    int blockNum = 1;

    for(int i = a->order - 1; i >= 0; i--){
        if(i > n)
            stride *= a->dimSize[i];
        else if(i < n)
            blockNum *= a->dimSize[i];
    }

    int cudaGrids[3];
    int cudaBlocks[3];

    int devIDBackup = 0;
    ProtectCudaDev(a->devID, devIDBackup);

    if (a->dataType == DEFAULT_DTYPE){
        if(stride > 1){
            GDevs.GetCudaThread2D(a->devID, stride * blockNum, blockSize, MAX_INT, cudaGrids, cudaBlocks);
            if(beta == (DTYPE)1.0F)
                KernelAddWithCol<DTYPE, false> <<<dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1])>>>
                                                ((DTYPE*)a->data, (DTYPE*)b->data, (DTYPE*)c->data, 
                                                  blockSize, stride, blockSize * stride, blockNum, beta);
            else
                KernelAddWithCol<DTYPE, true>  <<<dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1])>>>
                                                ((DTYPE*)a->data, (DTYPE*)b->data, (DTYPE*)c->data, 
                                                  blockSize, stride, blockSize * stride, blockNum, beta);
        }
        else if(stride == 1){
            GDevs.GetCudaThread2D(a->devID, blockSize, blockNum, MAX_INT, cudaGrids, cudaBlocks);
            if(beta == (DTYPE)1.0F)
                KernelAddWithRow<DTYPE, false> <<<dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1])>>>
                                                ((DTYPE*)a->data, (DTYPE*)b->data, (DTYPE*)c->data, 
                                                  blockNum, blockSize, beta);
            else
                KernelAddWithRow<DTYPE, true>  <<<dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1])>>>
                                                ((DTYPE*)a->data, (DTYPE*)b->data, (DTYPE*)c->data, 
                                                  blockNum, blockSize, beta);
        }
        else{
            ShowNTErrors("Something is wrong!");
        }
    }
    else if (a->dataType == X_FLOAT16) {
#ifdef HALF_PRECISION
        half beta1 = __float2half(beta);
        if (stride > 1) {
            GDevs.GetCudaThread2D(a->devID, stride * blockNum, blockSize, MAX_INT, cudaGrids, cudaBlocks);
            if (beta == (DTYPE)1.0F)
                KernelAddWithCol<__half, false> <<<dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1])>>>
                                                 ((__half*)a->data, (__half*)b->data, (__half*)c->data,
                                                   blockSize, stride, blockSize * stride, blockNum, beta1);
            else
                KernelAddWithCol<__half, true> <<<dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1])>>>
                                                ((__half*)a->data, (__half*)b->data, (__half*)c->data,
                                                  blockSize, stride, blockSize * stride, blockNum, beta1);
        }
        else if (stride == 1) {
            GDevs.GetCudaThread2D(a->devID, blockSize, blockNum, MAX_INT, cudaGrids, cudaBlocks);
            if (beta == (DTYPE)1.0F)
                KernelAddWithRow<__half, false> <<<dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1])>>>
                                                 ((__half*)a->data, (__half*)b->data, (__half*)c->data,
                                                   blockNum, blockSize, beta1);
            else
                KernelAddWithRow<__half, true> <<<dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1])>>>
                                                ((__half*)a->data, (__half*)b->data, (__half*)c->data,
                                                  blockNum, blockSize, beta1);
        }
        else {
            ShowNTErrors("Something is wrong!");
        }
#else
        ShowNTErrors("Recompile the code with HALF_PRECISION!");
#endif
    }
    else {
        ShowNTErrors("TODO!");
    }

    BacktoCudaDev(a->devID, devIDBackup);
}

#endif

} // namespace nts(NiuTrans.Tensor)

