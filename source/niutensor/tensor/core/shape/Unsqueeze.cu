#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
* Copyright (C) 2017, Natural Language Processing Lab, Northeastern University.
* All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-24
*/

#include "../../XDevice.h"
#include "../../XTensor.h"
#include "Unsqueeze.h"
#include "Unsqueeze.cuh"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef  USE_CUDA

/*
insert a dimension by copying the blocks for n times (where n is the size of the inerted dimension)
>> s - pointer to the source data array
>> blockSize - size of a block

>> totalSize - total size of the blocks (i.e., blockSIze * n)
>> t - pointer to the target data array
>> n - number of blocks to copy data
*/
template<class T>
__global__
void KernelUnsqueezeFlat(void * s, int blockSize, int totalSize, void * t, int n)
{
    /* index of data items */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= blockSize)
        return;

    T value = ((T*)s)[i];
    T * tData = (T*)t;

    __syncthreads();

    for (int k = i; k < totalSize; k += blockSize)
        tData[k] = value;
}

/*
insert a dimension by copying the blocks for n times (where n is the size of the inerted dimension)
>> s - pointer to the source data array
>> blockSize - size of a block

>> totalSize - total size of the blocks (i.e., blockSIze * n)
>> t - pointer to the target data array
>> n - number of blocks to copy data
*/
template<class T>
__global__
void KernelUnsqueezeFlatBigram(void * s, int blockSize, int totalSize, void * t, int n)
{
    /* index of data items */
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * 2;

    if (i >= blockSize)
        return;

    T value = ((T*)s)[i];
    T value2 = ((T*)s)[i + 1];
    T * tData = (T*)t;

    __syncthreads();

    for (int k = i; k < totalSize; k += blockSize){
        tData[k] = value;
        tData[k + 1] = value2;
    }
}

/*
insert a dimension by copying the blocks for n times (where n is the size of the inerted dimension)
>> s - pointer to the source data array
>> blockSize - size of a block

>> totalSize - total size of the blocks (i.e., blockSIze * n)
>> t - pointer to the target data array
>> n - number of blocks to copy data
*/
template<class T>
__global__
void KernelUnsqueezeFlat2D(void * s, int blockSize, int totalSize, void * t, int n)
{
    __shared__ T data[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ int offsets[MAX_CUDA_THREAD_NUM_PER_BLOCK];

    /* index of data items */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    /* index of data items */
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i >= blockSize || j >= n)
        return;

    if(threadIdx.y == 0)
        data[threadIdx.x] = ((T*)s)[i];
    if(threadIdx.x == 0)
        offsets[threadIdx.y] = blockSize * j;

    __syncthreads();

    ((T*)t)[offsets[threadIdx.y] + i] = data[threadIdx.x];
}

/*
insert a dimension by copying the blocks for n times (where n is the size of the inerted dimension)
>> s - pointer to the source data array
>> blockSize - size of a block
>> blockNum - number of the blocks
>> totalSize - total size of the blocks (i.e., blockSize * n)
>> t - pointer to the target data array
>> n - number of blocks to copy data
*/
template<class T>
__global__
void KernelUnsqueeze(void * s, int blockSize, int blockNum, int totalSize, void * t, int n)
{
    /* index of data items */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    /* block index */
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i >= blockSize || j >= blockNum)
        return;

    MTYPE offset = blockSize * j;
    T value = ((T*)s)[offset + i];
    T * tData = (T*)t + offset * n;

    __syncthreads();

    for (int k = i; k < totalSize; k += blockSize)
        tData[k] = value;
}

/*
insert a dimension by copying the blocks for n times (where n is the size of the inerted dimension)
This is special case where we actually copy a v-dimentional column vector by n times to form a v * n matrix
>> s - pointer to the source data array
>> rowNum - number of rows (i.e., dimension size of s)
>> colNum - number of columns (i.e., number of copies)
>> t - pointer to the target data array
*/
template<class T>
__global__
void KernelUnsqueezeByCol(void * s, int rowNum, int colNum, void * t)
{
    __shared__ T values[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ T * ts[MAX_CUDA_THREAD_NUM_PER_BLOCK];

    /* column index */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    /* row index */
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i >= colNum || j >= rowNum)
        return;

    if(threadIdx.x == 0){
        values[threadIdx.y] = ((T*)s)[j];
        ts[threadIdx.y] = (T*)t + colNum * j;
    }

    __syncthreads();

    ts[threadIdx.y][i] = values[threadIdx.y];
}

/*
insert a dimension by copying the blocks for n times (where n is the size of the inerted dimension)
This is special case where we actually copy a v-dimentional column vector by n times to form a v * n matrix
And a row is very big so that it occupies the cuda threads in a block
>> s - pointer to the source data array
>> rowNum - number of rows (i.e., dimension size of s)
>> colNum - number of columns (i.e., number of copies)
>> t - pointer to the target data array
*/
template<class T>
__global__
void KernelUnsqueezeByColBigRow(void * s, int rowNum, int colNum, void * t)
{
    __shared__ T value;
    __shared__ T * tData;

    /* column index */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    /* row index */
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i >= colNum || j >= rowNum)
        return;

    if (threadIdx.x == 0) {
        value = ((T*)s)[j];
        tData = (T*)t + colNum * j;
    }

    __syncthreads();

    tData[i] = value;
}

/*
insert a dimension by copying the blocks for x times (where x is the size of the inerted dimension)
>> a - input tensor
>> b - output tensor
>> dim - where to insert the dimension
>> dSize - size of the newly-inserted dimension
*/
void _CudaUnsqueeze(const XTensor * a, XTensor * b, int dim, int dSize)
{
    int blockSize = 1;
    int blockNumA = 1;
    int blockNumB = 1;
    for (int i = dim; i < a->order; i++)
        blockSize *= a->dimSize[i];

    blockNumA = a->unitNum / blockSize;
    blockNumB = b->unitNum / blockSize;

    CheckNTErrors((blockNumA * dSize == blockNumB), "Unmatched tensors!");;

    int cudaGrids[3];
    int cudaBlocks[3];

    int devIDBackup = 0;
    ProtectCudaDev(a->devID, devIDBackup);

    if (dim == b->order - 1) {
        GDevs.GetCudaThread2D(a->devID, dSize, blockNumA, MAX_INT, cudaGrids, cudaBlocks);

        if (a->dataType == X_FLOAT && b->dataType == X_FLOAT) {
            if (cudaBlocks[1] == 1)
                KernelUnsqueezeByColBigRow<float> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                                     (a->data, blockNumA, dSize, b->data);
            else
                KernelUnsqueezeByCol<float> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                               (a->data, blockNumA, dSize, b->data);
        }
        else if (a->dataType == X_INT && b->dataType == X_INT) {
            if (cudaBlocks[1] == 1)
                KernelUnsqueezeByColBigRow<int> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                                   (a->data, blockNumA, dSize, b->data);
            else
                KernelUnsqueezeByCol<int> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                             (a->data, blockNumA, dSize, b->data);
        }
        else if (a->dataType == X_FLOAT16 && b->dataType == X_FLOAT16) {
            if (cudaBlocks[1] == 1)
                KernelUnsqueezeByColBigRow<__half> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                                   (a->data, blockNumA, dSize, b->data);
            else
                KernelUnsqueezeByCol<__half> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                             (a->data, blockNumA, dSize, b->data);
        }
        else {
            ShowNTErrors("TODO!");
        }

        
    }
    else if(blockNumA > 1){
        GDevs.GetCudaThread2D(a->devID, blockSize, blockNumA, MAX_INT, cudaGrids, cudaBlocks);

        if (a->dataType == X_FLOAT && b->dataType == X_FLOAT) {
            KernelUnsqueeze<float> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                      (a->data, blockSize, blockNumA, blockSize * dSize, b->data, dSize);
        }
        else if (a->dataType == X_INT && b->dataType == X_INT) {
            KernelUnsqueeze<int> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                    (a->data, blockSize, blockNumA, blockSize * dSize, b->data, dSize);
        }
        else if (a->dataType == X_FLOAT16 && b->dataType == X_FLOAT16) {
            KernelUnsqueeze<half> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                    (a->data, blockSize, blockNumA, blockSize * dSize, b->data, dSize);
        }
        else {
            ShowNTErrors("TODO!");
        }
    }
    else if(blockNumA == 1 && blockSize < MAX_CUDA_THREAD_NUM_PER_BLOCK){
        GDevs.GetCudaThread2D(a->devID, blockSize, dSize, MAX_CUDA_THREAD_NUM_PER_BLOCK/4, cudaGrids, cudaBlocks);

        if (a->dataType == X_FLOAT && b->dataType == X_FLOAT) {
            KernelUnsqueezeFlat2D<float> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                          (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else if (a->dataType == X_INT && b->dataType == X_INT) {
            KernelUnsqueezeFlat2D<int> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                        (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else if (a->dataType == X_FLOAT16 && b->dataType == X_FLOAT16) {
            KernelUnsqueezeFlat2D<half> << <dim3(cudaGrids[0], cudaGrids[1]), dim3(cudaBlocks[0], cudaBlocks[1]) >> >
                                           (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else {
            ShowNTErrors("TODO!");
        }
    }
    else if(blockNumA == 1 && blockSize % 2 == 0){
        GDevs.GetCudaThread(a->devID, blockSize/2, cudaGrids, cudaBlocks);

        if (a->dataType == X_FLOAT && b->dataType == X_FLOAT) {
            KernelUnsqueezeFlatBigram<float> << <dim3(cudaGrids[0]), dim3(cudaBlocks[0]) >> >
                                                (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else if (a->dataType == X_INT && b->dataType == X_INT) {
            KernelUnsqueezeFlatBigram<int> << <dim3(cudaGrids[0]), dim3(cudaBlocks[0]) >> >
                                              (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else if (a->dataType == X_FLOAT16 && b->dataType == X_FLOAT16) {
            KernelUnsqueezeFlatBigram<half> << <dim3(cudaGrids[0]), dim3(cudaBlocks[0]) >> >
                                               (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else {
            ShowNTErrors("TODO!");
        }
    }
    else if(blockNumA == 1){
        GDevs.GetCudaThread(a->devID, blockSize, cudaGrids, cudaBlocks);

        if (a->dataType == X_FLOAT && b->dataType == X_FLOAT) {
            KernelUnsqueezeFlat<float> << <dim3(cudaGrids[0]), dim3(cudaBlocks[0]) >> >
                                          (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else if (a->dataType == X_INT && b->dataType == X_INT) {
            KernelUnsqueezeFlat<int> << <dim3(cudaGrids[0]), dim3(cudaBlocks[0]) >> >
                                        (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else if (a->dataType == X_FLOAT16 && b->dataType == X_FLOAT16) {
            KernelUnsqueezeFlat<half> << <dim3(cudaGrids[0]), dim3(cudaBlocks[0]) >> >
                                         (a->data, blockSize, blockSize * dSize, b->data, dSize);
        }
        else {
            ShowNTErrors("TODO!");
        }
    }
    else{
        ShowNTErrors("Something is wrong!");
    }

    BacktoCudaDev(a->devID, devIDBackup);
}

#endif // USE_CUDA
} // namespace nts(NiuTrans.Tensor)