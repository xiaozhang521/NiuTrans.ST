#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northeastern University.
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: Xu Chen (email: hello_master1954@163.com) 2018-11-30
 */

#include "CopyIndexed.cuh"
#include "../../XDevice.h"
#include "../../XUtility.h"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/*
copy selected sub-tensors where indeces are kept in tensors (kenerl version)

>> s - the source tensor
>> t - the target tensor
>> dim - the leading dimension to define "sub-tensors"
         e.g., for a tensor of size (3, 2, 4) and dim = 2, 
         we have 4 sub-tensors of size (3, 2)
>> srcIndex - the tensor to save the index of the source sub-tensors
>> tgtIndex - the tensor to save the index of the target sub-tensors
>> copyNum - number of the sub-tensors we copy for each source index, 
             e.g., for srcIndex = [1,4] and copyNum = 2,
             we actually copy the source sub-tensors 1, 2, 4, 5
*/
template <class T>
__global__
void KernelCopyIndexed(T * sData, T * tData, int * sIndex, int * tIndex,
 
                       int blockNum, int blockSizeSrc, int blockSizeTgt, 
                       int stride, int indexSize, int copyNum)
{
    __shared__ T * sp[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ T * tp[MAX_CUDA_THREAD_NUM_PER_BLOCK];

    /* block id */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    /* offset in each block */
    int offset = blockDim.y * blockIdx.y + threadIdx.y;

    if(i >= blockNum * indexSize * copyNum || offset >= stride)
        return;

    int realIndexSize = indexSize * copyNum;

    int realBlockNum = i / realIndexSize;
    int realIndex = i % realIndexSize;

    int realSrcIndex = sIndex[realIndex / copyNum] + realIndex % copyNum;
    int realTgtIndex = tIndex[realIndex / copyNum] + realIndex % copyNum;

    if(threadIdx.y == 0){
        sp[threadIdx.x] = sData + realBlockNum * blockSizeSrc + realSrcIndex * stride;
        tp[threadIdx.x] = tData + realBlockNum * blockSizeTgt + realTgtIndex * stride;
    }

    __syncthreads();

    T * s = sp[threadIdx.x];
    T * t = tp[threadIdx.x];

    t[offset] = s[offset];
}

/*
copy selected sub-tensors where indeces are kept in tensors

>> s - the source tensor
>> t - the target tensor
>> dim - the leading dimension to define "sub-tensors"
         e.g., for a tensor of size (3, 2, 4) and dim = 2, 
         we have 4 sub-tensors of size (3, 2)
>> srcIndex - the tensor to save the index of the source sub-tensors
>> tgtIndex - the tensor to save the index of the target sub-tensors
>> copyNum - number of the sub-tensors we copy for each source index, 
             e.g., for srcIndex = [1,4] and copyNum = 2,
             we actually copy the source sub-tensors 1, 2, 4, 5
*/
void _CudaCopyIndexed(const XTensor * s, XTensor * t, int dim,
                      const XTensor * srcIndex, const XTensor * tgtIndex,
                      int copyNum)
{
    int devID = s->devID;
    int order = s->order;
    int indexSize = srcIndex->unitNum;

    int blockNum = 1;
    int stride = 1;
    int blockSizeSrc = 1;
    int blockSizeTgt = 1;

    for (int i = 0; i < dim; i++)
        blockNum *= s->GetDim(i);
    
    for (int i = dim + 1; i < order; i++)
        stride *= s->GetDim(i);

    blockSizeSrc = stride * s->GetDim(dim);
    blockSizeTgt = stride * t->GetDim(dim);

    int cudaGrids[3];
    int cudaBlocks[3];

    int devIDBackup;
    ProtectCudaDev(devID, devIDBackup);

    GDevs.GetCudaThread2D(devID, blockNum * indexSize * copyNum, stride, MAX_INT, cudaGrids, cudaBlocks);

    dim3 blocks(cudaGrids[0], cudaGrids[1]);
    dim3 threads(cudaBlocks[0], cudaBlocks[1]);

    if (s->dataType == DEFAULT_DTYPE && t->dataType == DEFAULT_DTYPE) {
        DTYPE * sData = (DTYPE*)s->data;
        DTYPE * tData = (DTYPE*)t->data;

        int * sIndex = (int*)srcIndex->data;
        int * tIndex = (int*)tgtIndex->data;

        KernelCopyIndexed<<<blocks, threads >>>(sData, tData, sIndex, tIndex,
                                                blockNum, blockSizeSrc, blockSizeTgt,
                                                stride, indexSize, copyNum);
    }
    else if (s->dataType == X_FLOAT16 && t->dataType == X_FLOAT16) {
        half * sData = (half*)s->data;
        half * tData = (half*)t->data;

        int * sIndex = (int*)srcIndex->data;
        int * tIndex = (int*)tgtIndex->data;
        KernelCopyIndexed<<<blocks, threads>>>(sData, tData, sIndex, tIndex,
                                               blockNum, blockSizeSrc, blockSizeTgt,
                                               stride, indexSize, copyNum);
    }
    else {
        ShowNTErrors("Unsupported dataType!");
    }
    BacktoCudaDev(devID, devIDBackup);

}

#endif // USE_CUDA

} // namespace nts(NiuTrans.Tensor)