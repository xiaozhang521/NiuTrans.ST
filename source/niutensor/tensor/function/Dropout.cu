#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northeastern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: Xu Chen (email: hello_master1954@163.com) 2018-09-12
 */

#include "Dropout.h"
#include "Dropout.cuh"
#include "Loss.cuh"
#include "../XDevice.h"

#ifdef USE_CUDA

// the CUDA stuff
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#endif

namespace nts{ // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/* 
dropout function (Cuda kernel) 
>> x - input data pointer
>> y - output data pointer
>> m - mask indicator to set zero 
>> s - the scale factor
>> size - size of input/output
*/
__global__ 
void KernelDropoutCompute(DTYPE * x, DTYPE * y, DTYPE * m, DTYPE s, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        y[i] = x[i] * m[i] * s;
    }
}

/*
dropout function (Cuda version)
>> x - input tensor
>> y - output tensor
>> mask - mask tensor to set 0
>> scaleFactor - the scale factor
*/
void _CudaDropout(const XTensor * x, XTensor * y, const XTensor * mask, DTYPE scaleFactor)
{
    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){

        CheckNTErrors(!x->isSparse && !y->isSparse, "the activation function (rectify) does not support sparse matrices.");
        CheckNTErrors(x->unitNum && y->unitNum, "we require two vectors with the same length.");

        int gridSize[3], blockSize[3];

        GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

        int devIDBackup;
        ProtectCudaDev(x->devID, devIDBackup);

        KernelDropoutCompute<<<dim3(gridSize[0]), dim3(blockSize[0])>>>((DTYPE*)x->data, (DTYPE*)y->data, (DTYPE*)mask->data, scaleFactor, x->unitNum);

        BacktoCudaDev(x->devID, devIDBackup);
    }
    else
        ShowNTErrors("TODO!");
}

/* 
backward computation of dropout function (Cuda kernel)

dE/dx = dE/dy * dy/dx

>> dedy - dE/dy
>> dedx - dE/dx
>> m - mask indicator to set zero 
>> s - the scale factor
>> size - size of input/output
*/
__global__
void KernelDropoutBackward(DTYPE * dedy, DTYPE * dedx, 
                           DTYPE * m, DTYPE s, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        dedx[i] = dedy[i] * m[i] * s;
    }
}

/* 
backward computation of dropout function (Cuda version)

dE/dx = dE/dy * dy/dx

>> y - output of the dropout function
>> x - input of the dropout function
>> dedy - dE/dy
>> dedx - dE/dx
>> mask - mask tensor to set 0
>> scaleFactor - the scale factor
*/
void _CudaDropoutBackward(const XTensor * y, const XTensor * x,
                          const XTensor * dedy, XTensor * dedx,
                          const XTensor * mask, DTYPE scaleFactor)
{
    int gridSize[3], blockSize[3];

    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){
        GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

        int devIDBackup;
        ProtectCudaDev(x->devID, devIDBackup);

        /* dE/ds = dE/dy * dy/ds */
        KernelDropoutBackward<<<dim3(gridSize[0]),dim3(blockSize[0])>>>
                              ((DTYPE*)dedy->data, (DTYPE*)dedx->data, 
                               (DTYPE*)mask->data, scaleFactor, x->unitNum);

        BacktoCudaDev(x->devID, devIDBackup);
    }
    else
        ShowNTErrors("TODO!");
}

#endif

} // namespace nts(NiuTrans.Tensor)