#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northeastern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: Yuhao Zhang (yoohao.zhang@gmail.com) 2023-09-21
 */

#include "GELU.cuh"
#include "../XDevice.h"

namespace nts{ // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/* 
hard rectify computation (Cuda kernel) 
rectify   : y =  x    if x >= 0
                 0    if x < 0
>> input - input tensor
>> output - output tensor
>> size - size of input/output
*/
template<class T>
__global__ 
void KernelGELU(T * x, T * y, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size) {
        T x_val = x[i];
        // 1.414213562373095145475 1.128379167095512558561
        //T sqrt_2_over_pi = (T)sqrt(2.0 / M_PI);
        //T sqrt_2_over_pi = 1.414213562373095145475 * (1.0 / (T)sqrt(M_PI));
        T sqrt_2_over_pi = 0.707106781186547572737 * 1.128379167095512558561;
        T scale = (T)0.5;
        T tanh_val = tanh(sqrt_2_over_pi * ((T)0.044715 * x_val * x_val * x_val + x_val));
        y[i] = scale * (x_val * tanh_val + x_val);
    }
}

/*
rectify function y = max(0, x)
>> x - input tensor
>> y - result
*/
void _CudaGELU(const XTensor * x, XTensor * y)
{
    int gridSize[3], blockSize[3];

    GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    if (x->dataType == DEFAULT_DTYPE) {  
        KernelGELU<<<dim3(gridSize[0]), dim3(blockSize[0])>>>
                        ((DTYPE*)x->data, (DTYPE*)y->data, x->unitNum);
    }
    else if (x->dataType == X_FLOAT16) {
#ifdef HALF_PRECISION
        KernelRectify<<<dim3(gridSize[0]), dim3(blockSize[0]) >> >
                        ((__half*)x->data, (__half*)y->data, x->unitNum);
#else
        ShowNTErrors("Recompile the code with HALF_PRECISION!");
#endif
    }
    else {
        // TODO!!
        ShowNTErrors("TODO!");
    }

    BacktoCudaDev(x->devID, devIDBackup);
}

/* 
rectify backward computation of dE/dx (Cuda kernel)

dy/dx =  1    if x >= 0
         0    otherwise

>> dedy - dE/dy
>> dedx - dE/dx
>> x - input of the function
>> size - size of output/input
*/
template<class T>
__global__ 
void KernelGELUBackward(T * dedy, T * dedx, T * x, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size) {
        T x_val = x[i];
        T sqrt_2_over_pi = (T)sqrt(2.0 / M_PI);
        T a = (T)0.044715;
        T b = (T)0.134145;
        T c = (T)1.0;
        T tanh_val = tanh(sqrt_2_over_pi * (x_val + a * x_val * x_val * x_val));
        T derivative = 0.5 * (c + tanh_val) + 0.5 * x_val * (c - tanh_val * tanh_val) * sqrt_2_over_pi * (c + b * x_val * x_val);
        dedx[i] = dedy[i] * derivative;
    }
}


/*
backward computation (Cuda version)

dE/dx = dE/dy * dy/dx

rectify  : y =  s    if s >= 0
                0    if s < 0

   and dy/ds =  1    if s >= 0
                0    otherwise

>> y - output of the rectify function
>> x - input of the rectify function
>> dedy - dE/dy
>> dedx - dE/dx
*/
void _CudaGELUBackward(XTensor * y, XTensor * x, 
                          XTensor * dedy, XTensor * dedx)
{
    int gridSize[3], blockSize[3];

    GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    /* dE/ds = dE/dy * dy/ds */
    if (x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE) {   
        KernelGELUBackward<<<dim3(gridSize[0]),dim3(blockSize[0])>>>
                              ((DTYPE*)dedy->data, 
                               (DTYPE*)dedx->data,
                               (DTYPE*)x->data, 
                                x->unitNum);
    }
    else if (x->dataType == X_FLOAT16 && y->dataType == X_FLOAT16) {
#ifdef HALF_PRECISION
        KernelGELUBackward<<<dim3(gridSize[0]), dim3(blockSize[0]) >> >
                              ((__half*)dedy->data,
                               (__half*)dedx->data,
                               (__half*)x->data,
                                x->unitNum);
#else
        ShowNTErrors("Recompile the code with HALF_PRECISION!");
#endif
    }
    else {
        // TODO!!
        ShowNTErrors("TODO!");
    }

    BacktoCudaDev(x->devID, devIDBackup);
}

#endif

} // namespace nts(NiuTrans.Tensor)