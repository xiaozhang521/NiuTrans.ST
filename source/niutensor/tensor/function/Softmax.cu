#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
* Copyright (C) 2017, Natural Language Processing Lab, Northeastern University.
* All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-26
*/

#include "Softmax.h"
#include "Softmax.cuh"
#include "Loss.cuh"
#include "../core/reduce/ReduceSum.h"
#include "../core/arithmetic/Multiply.h"
#include "../core/arithmetic/MultiplyDim.h"
#include "../core/shape/Unsqueeze.h"
#include "../core/shape/IsSameShaped.h"
#include "../core/arithmetic/Sum.h"
#include "../XDevice.h"
#include "../XUtility.h"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/*
softmax y = e^x / \sum_{i} e^{x_i} (Cuda version)
>> x - x vector
>> y - result
>> leadDim - leading dimension (along which we perform reduction)
*/
void _CudaSoftmax(const XTensor * x, XTensor * y, int leadDim)
{
    ShowNTErrors("You should call Softmax instead!");
}

/* 
softmax forward computation (Cuda kernel)

given a data block, 
for each column j, let y_{i,j} and x_{i,j} are the y
and state value for the i-th element of column j. We have

 y_{i,j} = e^{x_{i,j}-max_j} / \sum_{i} e^{x_{i,j}-max_j}

>> x - x tensor
>> max - the max value for each column j
>> sum - \sum_{i} e^{s_{i,j}) for each column j
>> y - y tensor
>> stride - number of items we go over when move to the next step alone the leading dimension
>> strideNum - size of the leading dimension in a block
>> blockSize - size of a block (i.e., stride * strideNum)
>> blockNum - number of blocks
>> strideSizeTotal - stride * blockNum
*/
__global__ 
void KernelSoftmaxComputeTensor(DTYPE * x, DTYPE * max, DTYPE * sum, DTYPE * y, int stride, int strideNum, int blockSize, int blockNum, int strideSizeTotal)
{
    __shared__ DTYPE xSum[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ DTYPE xMax[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ int i2[MAX_CUDA_THREAD_NUM_PER_BLOCK];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    /* we keep the sum and max number in the shared memory for each column */
    if(threadIdx.y == 0){
        xSum[threadIdx.x] = sum[i];
        xMax[threadIdx.x] = max[i];
        i2[threadIdx.x] = i % stride;
    }

    /* synchronize to make sure the values of max and sum are loaded */
    __syncthreads();

    if(i < strideSizeTotal && j < strideNum){
        int offset = int(i / stride) * blockSize + j * stride + i2[threadIdx.x];
        DTYPE r = exp(x[offset] - xMax[threadIdx.x])/xSum[threadIdx.x];
        if (r >(DTYPE)1.0F)
            r = (DTYPE)1.0F;
        else if (r < 0)
            r = 0;
        y[offset] = r;

    }
}

/* 
softmax forward computation (Cuda kernel)
This is for float16 computation

given a data block, 
for each column j, let y_{i,j} and x_{i,j} are the y
and state value for the i-th element of column j. We have

 y_{i,j} = e^{x_{i,j}-max_j} / \sum_{i} e^{x_{i,j}-max_j}

>> x - x tensor
>> max - the max value for each column j
>> sum - \sum_{i} e^{s_{i,j}) for each column j
>> y - y tensor
>> stride - number of items we go over when move to the next step alone the leading dimension
>> strideNum - size of the leading dimension in a block
>> blockSize - size of a block (i.e., stride * strideNum)
>> blockNum - number of blocks
>> strideSizeTotal - stride * blockNum
*/
__global__ 
void KernelSoftmaxComputeTensor(__half * x, __half * max, __half * sum, __half * y, int stride, int strideNum, int blockNum)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    __shared__ int i2[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ int blockSize;

#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    __shared__ __half xSum[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ __half xMax[MAX_CUDA_THREAD_NUM_PER_BLOCK];

    /* we keep the sum and max number in the shared memory for each column */
    if(threadIdx.y == 0){
        xSum[threadIdx.x] = sum[i];
        xMax[threadIdx.x] = max[i];
        i2[threadIdx.x] = i % stride;
        blockSize = stride * strideNum;
    }
#else
    __shared__ DTYPE xSum[MAX_CUDA_THREAD_NUM_PER_BLOCK];
    __shared__ DTYPE xMax[MAX_CUDA_THREAD_NUM_PER_BLOCK];

    /* we keep the sum and max number in the shared memory for each column */
    if(threadIdx.y == 0){
        xSum[threadIdx.x] = __half2float(sum[i]);
        xMax[threadIdx.x] = __half2float(max[i]);
        i2[threadIdx.x] = i % stride;
        blockSize = stride * strideNum;
    }
#endif

    /* synchronize to make sure the values of max and sum are loaded */
    __syncthreads();

    if(i < stride * blockNum && j < strideNum){
        int offset = int(i / stride) * blockSize + j * stride + i2[threadIdx.x];
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
        y[offset] = __hdiv(hexp(x[offset] - xMax[threadIdx.x]), xSum[threadIdx.x]);
#else
        y[offset] = __float2half(exp(__half2float(x[offset]) - xMax[threadIdx.x])/xSum[threadIdx.x]);
#endif
    }
}

/*
use PTX code to broadcast float data
*/
__device__ __forceinline__ 
float broadcast(float input)
{
    float output;
    asm(
        "{"
        "shfl.sync.idx.b32 %0,%1,0x0,0x1f,0xffffffff;"
        "}"
        :"=f"(output) : "f"(input)
    );
    return output;
}

/*
use warp broadcast to optimize softmax computing
*/
__global__
void KernelSoftmaxComputeTensorUseBroadcast(DTYPE * input, DTYPE * max, DTYPE * sum, DTYPE * output, 
                                            int stride, int strideNum, int blockNum)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int i2 = j % stride;
    int blockSize = stride * strideNum;

    if (j < stride * blockNum) {
        DTYPE sumData, maxData;
        if (i % 32 == 0) {
            sumData = sum[j];
            maxData = max[j];
        }
        sumData = broadcast(sumData);
        maxData = broadcast(maxData);
        if (i < strideNum){
            int offset = int(j / stride) * blockSize + i * stride + i2;
            DTYPE r = exp(input[offset] - maxData) / sumData;
            if (r > (DTYPE)1.0F)
                r = (DTYPE)1.0F;
            else if (r < 0)
                r = 0;
            output[offset] = r;
        }
    }
}

/*
softmax y = e^x / \sum_{i} e^{x_i} (Cuda version)
>> x - x vector
>> y - result
>> leadDim - leading dimension (along which we perform reduction)
>> sum - \sum_{i} e^{x_i}
>> max - \max_{i} e^{x_i}
*/
void _CudaSoftmaxSumMax(const XTensor * x, XTensor * y, int leadDim, XTensor * sum, XTensor * max)
{
    CheckNTErrors((x->devID >= 0), "Forward computation of softmax must be run on GPUs.");
    CheckNTErrors((x->devID == y->devID), "Tensors used in softmax are not on the same GPU.");
    CheckNTErrors((_IsSameShaped(x, y)), "Input tensors must be of the same size!");

    int dimensionSize = y->dimSize[leadDim];
    int stride = 1;
    int blockSize = 1;
    int blockNum = 1;

    for(int i = leadDim + 1; i < y->order; i++)
        stride *= y->dimSize[i];
    blockSize = stride * dimensionSize;
    blockNum = y->unitNum / blockSize;

    int cudaGridSize[3];
    int cudaBlockSize[3];

    if (leadDim != 0 || dimensionSize <= 10){
        /* allocate thread num for old function */
        GDevs.GetCudaThread2D(x->devID, stride * blockNum, dimensionSize, MAX_INT, cudaGridSize, cudaBlockSize);
    }
    else {
        /* allocate thread num for new function */
        GDevs.GetCudaThread2D(x->devID, dimensionSize, stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);
        if (cudaBlockSize[0] < 32) {
            /* use at least a warp */
            cudaBlockSize[0] = 32;

            if (cudaBlockSize[1] > 32) {
                cudaGridSize[1] = int(ceil(float(stride * blockNum) / 32));
                cudaBlockSize[1] = 32;
            }
        }
    }
    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){
        if (leadDim != 0 || dimensionSize <= 10) {
            KernelSoftmaxComputeTensor <<< dim3(cudaGridSize[0], cudaGridSize[1]), dim3(cudaBlockSize[0], cudaBlockSize[1]) >>>
                                         ((DTYPE*)x->data, (DTYPE*)max->data, (DTYPE*)sum->data, (DTYPE*)y->data,
                                           stride, dimensionSize, stride * dimensionSize, blockNum, stride * blockNum);
        }
        else {
            KernelSoftmaxComputeTensorUseBroadcast <<< dim3(cudaGridSize[0], cudaGridSize[1]), dim3(cudaBlockSize[0], cudaBlockSize[1]) >>>
                                                     ((DTYPE*)x->data, (DTYPE*)max->data, (DTYPE*)sum->data, (DTYPE*)y->data,
                                                       stride, dimensionSize, blockNum);
        }
    }
    else if(x->dataType == X_FLOAT16 && y->dataType == X_FLOAT16){
        KernelSoftmaxComputeTensor <<< dim3(cudaGridSize[0], cudaGridSize[1]), dim3(cudaBlockSize[0], cudaBlockSize[1]) >>>
                                     ((__half*)x->data, (__half*)max->data, (__half*)sum->data, (__half*)y->data, 
                                       stride, dimensionSize, blockNum);
    }
    else{
        ShowNTErrors("TODO!");
    }

    BacktoCudaDev(x->devID, devIDBackup);
}

/*
backward computation for dense matrics with default data type

dE/ds = dE/dy * dy/dx

    softmax: y_i = e^{x_i} / \sum_{k} e^{x_k}

       dy_i/dx_j = y_i * (\delta(i,j) - y_j)

for cross-entropy error function,

         dE/dy_i = -gold_i / y_i
then
         dE/dx_j = -gold_j + y_j

See more details in SoftmaxBackward

>> gold - gold standard to measure error (or loss)
>> y - y of the function
>> x - x of the function
>> dedy - dE/dy
>> dedx - dE/dx
>> lossName - type of loss function, e.g., cross entropy
>> leadDim - leading dimension (along which we perform reduction)
*/
void _CudaSoftmaxBackward(XTensor * gold, XTensor * y, XTensor * x, 
                          XTensor * dedy, XTensor * dedx,
                          XTensor * padding, int leadDim,
                          LOSS_FUNCTION_NAME lossName)
{
    int n = leadDim < 0 ? y->order - 1 : leadDim;

    CheckNTErrors((x->devID >= 0), "Backward computation of log softmax must be run on GPUs.");
    CheckNTErrors((x->devID == y->devID), "Matrices used in log softmax are not on the same GPU.");
    CheckNTErrors((y->order >= 1), "Empty tensor!");

    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){
        
        CheckNTErrors((lossName == CROSSENTROPY || 
                       lossName == SQUAREDERROR || 
                       lossName == ONEHOTERROR || 
                       lossName == NOLOSS),
                       "Unknown loss function.");

        if(lossName == CROSSENTROPY || lossName == SQUAREDERROR){
            _Sum(y, gold, dedx, -1.0F);
            if(padding != NULL) {
                int paddingOrder = padding->order;
                int * paddingDims = new int[paddingOrder];
                memcpy(paddingDims, padding->dimSize, padding->order * sizeof(int));
                padding->Reshape(padding->unitNum);

                int order = dedx->order;
                int * dims = new int[order];
                memcpy(dims, dedx->dimSize, dedx->order * sizeof(int));
                dedx->Reshape(dedx->unitNum/dedx->GetDim(n), dedx->GetDim(n));
                _MultiplyDimMe(dedx, padding, 0);

                padding->Reshape(paddingOrder, paddingDims);
                dedx->Reshape(order, dims);

                delete[] paddingDims;
                delete[] dims;
            }
        }
        else if(lossName == ONEHOTERROR){
            ShowNTErrors("TODO!");
        }
        else if(lossName == NOLOSS){
            /*
            for softmax: 
            y_i = e^{x_i} / \sum_{k} e^{x_k}
            we have
            dy_i/ds_j = y_i * (\delta(i,j) - y_j)
            Then
            dE/dx_j = \sum_i dE/dy_i * dy_i/dx_j
                    = \sum_i dE/dy_i * y_i * (\delta(i,j) - y_j) 
                    = dE/dy_j * y_j - y_j * \beta
                    = y_j * (dE/dy_j - \beta)
            where
            \beta = \sum_i (dE/dy_i * y_i) 
            */

            int * dimSize = new int[y->order];
            for(int i = 0; i < y->order; i++){
                if(i < leadDim)
                    dimSize[i] = y->dimSize[i];
                else if(i > leadDim)
                    dimSize[i - 1] = y->dimSize[i];
            }

            /* make a matrix of the same size as the y (i.e., y) */
            XTensor * ytmp = NewTensor(y);

            /* make a matrix to keep \beta */
            XTensor * beta = NewTensorV2(y->order - 1, dimSize, y->dataType, y->denseRatio, y->devID, y->mem);

            /* \beta = \sum_i (dE/dy_i * y_i) */
            _Multiply(dedy, y, ytmp, 0, 0);
            _ReduceSum(ytmp, beta, leadDim);

            /* ytmp = dE/dy_j - \beta */
            _Unsqueeze(beta, ytmp, leadDim, y->dimSize[leadDim]);
            _Sum(dedy, ytmp, ytmp, -1.0F);

            /* dE/ds_j = y_j * ytmp = y_j * (dE/dy_j - \beta) */
            _Multiply(y, ytmp, dedx, 0, 0);

            delete[] dimSize;
            delete ytmp;
            delete beta;
        }
        else{
            ShowNTErrors("TODO!");
        }
    }
    else
        ShowNTErrors("TODO!");

    BacktoCudaDev(x->devID, devIDBackup);
}

#endif

} // namespace nts(NiuTrans.Tensor)
