#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northeastern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-25
*/

#include "HardTanH.h"
#include "HardTanH.cuh"
#include "../XDevice.h"

namespace nts{ // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/* 
hard tanh forward computation (Cuda kernel) 
y =  1    if x > 1
     x    if -1 <= x <= 1
    -1    if x < -1
>> x - input data array
>> y - output data array
>> size - size of input/output
*/
template <class T>
__global__ 
void KernelHardtanhCompute(T * x, T * y, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        T p = x[i];
        if (p >(T)1.0)
            p = (T)1.0;
        else if (p < (T)-1.0)
            p = (T)-1.0;
        y[i] = p;
    }
}

/*
hard tanh forward computation (Cuda version) 
y =  1    if x > 1
     x    if -1 <= x <= 1
    -1    if x < -1
>> x - input tensor
>> y - output tensor
*/
void _CudaHardTanH(const XTensor * x, XTensor * y)
{
    CheckNTErrors(!x->isSparse && !y->isSparse, 
                  "The hard tanh activation function does not support sparse tensors.");

    int gridSize[3], blockSize[3];

    GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){
        KernelHardtanhCompute<<<dim3(gridSize[0]), dim3(blockSize[0])>>>((DTYPE*)x->data, (DTYPE*)y->data, x->unitNum);
    }
    else if (x->dataType == X_FLOAT16 && y->dataType == X_FLOAT16) {
#ifdef HALF_PRECISION
        KernelHardtanhCompute<<<dim3(gridSize[0]), dim3(blockSize[0])>>>((__half *)x->data, (__half *)y->data, x->unitNum);
#else
        ShowNTErrors("Recompile the code with HALF_PRECISION!");
#endif
    }
    else {
        //TODO!
        ShowNTErrors("TODO!");
    }

    BacktoCudaDev(x->devID, devIDBackup);
}

/* 
hard tanh backward computation of dE/dx (Cuda kernel)

dy/dx = 1     if -1 <= x <= 1
        0     otherwise

>> dedy - dE/dy
>> dedx - dE/dx
>> x - x of the function
>> size - size of y/x
*/
template <class T>
__global__ 
void KernelHardtanhBackward(T * dedy, T * dedx, T * x, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        T s = x[i];
        if(s > (T)1.0 || s < (T)-1.0)
            dedx[i] = 0;
        else
            dedx[i] = dedy[i];
    }
}

/*
backward computation (Cuda version)

dE/dx = dE/dy * dy/dx

hard tanh: y =  1    if x > 1
                x    if -1 <= x <= 1
               -1    if x< -1

   and dy/dx =  1    if -1 <= x <= 1
                0    otherwise

>> y - output of the hardtanh function
>> x - input of the hardtanh function
>> dedy - dE/dy
>> dedx - dE/dx
*/
void _CudaHardTanHBackward(XTensor * y, XTensor * x, 
                           XTensor * dedy, XTensor * dedx)
{
    int gridSize[3], blockSize[3];

    GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){
        /* dE/dx = dE/dy * dy/dx */
        KernelHardtanhBackward<<<dim3(gridSize[0]),dim3(blockSize[0])>>>
                                ((DTYPE*)dedy->data,
                                (DTYPE*)dedx->data,
                                (DTYPE*)x->data,
                                 x->unitNum);
    }
    else if (x->dataType == X_FLOAT16 && y->dataType == X_FLOAT16) {
#ifdef HALF_PRECISION
        /* dE/dx = dE/dy * dy/dx */
        KernelHardtanhBackward<<<dim3(gridSize[0]), dim3(blockSize[0])>>>
                               ((half*)dedy->data,
                                (half*)dedx->data,
                                (half*)x->data,
                                 x->unitNum);
#else
        ShowNTErrors("Recompile the code with HALF_PRECISION!");
#endif
    }
    else {
        ShowNTErrors("Unsupported dataType!");
    }

    BacktoCudaDev(x->devID, devIDBackup);
}

#endif

} // namespace nts(NiuTrans.Tensor)