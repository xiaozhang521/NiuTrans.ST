#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northeastern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-25
 */

#include "Sigmoid.h"
#include "Sigmoid.cuh"
#include "Loss.cuh"
#include "../loss/CrossEntropy.cuh"
#include "../XDevice.h"

#ifdef USE_CUDA

// the CUDA stuff
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#endif

namespace nts{ // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/* 
sigmoid function y = 1/(1+exp(-x))  (Cuda kernel) 
>> x - input data pointer
>> y - output data pointer
>> size - size of input/output
*/
__global__ 
void KernelSigmoidCompute(DTYPE * x, DTYPE * y, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        y[i] = 1/(1+exp(-x[i]));
    }
}

/*
sigmoid function y = 1/(1+exp(-x)) (Cuda version)
>> x - input vector
>> y - result
*/
void _CudaSigmoid(const XTensor * x, XTensor * y)
{
    CheckNTErrors(!x->isSparse && !y->isSparse, "the activation function (rectify) does not support sparse matrices.");
    CheckNTErrors(x->unitNum && y->unitNum, "we require two vectors with the same length.");

    int gridSize[3], blockSize[3];

    GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

    int devIDBackup;
    ProtectCudaDev(x->devID, devIDBackup);

    KernelSigmoidCompute<<<dim3(gridSize[0]), dim3(blockSize[0])>>>((DTYPE*)x->data, (DTYPE*)y->data, x->unitNum);

    BacktoCudaDev(x->devID, devIDBackup);
}

/* 
sigmoid backward computation of dE/dx (Cuda kernel)

dE/ds = dE/dy * dy/dx

sigmoid: y = 1/(1+exp(-x))

   and dy/ds = y * (1 -y)

>> dedy - dE/dy
>> dedx - dE/ds
>> y - output of the function
>> x - input of the function
>> size - size of output/input
*/
__global__ 
void KernelSigmoidBackward(DTYPE * dedy, DTYPE * dedx, DTYPE * y, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        dedx[i] = dedy[i] * y[i] * ((DTYPE)1.0 - y[i]);
    }
}

/*
backward computation (Cuda version)

dE/ds = dE/dy * dy/dx

sigmoid: y = 1/(1+exp(-x))

   and dy/dx = y * (1 -y)

>> y - output of the function
>> x - input of the function
>> dedy - dE/dy
>> dedx - dE/dx
*/
void _CudaSigmoidBackward(XTensor * y, XTensor * x, 
                          XTensor * dedy, XTensor * dedx)
{
    int gridSize[3], blockSize[3];

    GDevs.GetCudaThread(y->devID, y->unitNum, gridSize, blockSize);

    int devIDBackup;
    ProtectCudaDev(y->devID, devIDBackup);

    /* dE/dx = dE/dy * dy/dx */
    KernelSigmoidBackward<<<dim3(gridSize[0]),dim3(blockSize[0])>>>
                            ((DTYPE*)dedy->data,
                            (DTYPE*)dedx->data,
                            (DTYPE*)y->data,
                            y->unitNum);

    BacktoCudaDev(x->devID, devIDBackup);
}

#endif // USE_CUDA

} // namespace nts(NiuTrans.Tensor)